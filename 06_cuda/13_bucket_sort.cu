
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void initialize(int *bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  bucket[i] = 0;
}

__global__ void reduction(int *bucket, int *key) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void makeoffset(int *bucket, int *tmp, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for(int j=1; j<range; j<<=1){
    tmp[i] = bucket[i];
    __syncthreads();
    bucket[i] += tmp[i-j];
    __syncthreads();
  }
}

__global__ void sort(int *bucket, int *key, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int bucket[5] = {8, 18, 32, 41, 50};
  for(int j=range-1; j>=0; j--){
    if(i>=bucket[j]) return;
    key[i] = j;
  }

}

int main() {
  int n = 50;
  int range = 5;
  int *key;
  hipMallocManaged(&key, n*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
  
  int *bucket;
  hipMallocManaged(&bucket, range*sizeof(int));
  initialize<<<1, range>>>(bucket);
  hipDeviceSynchronize();
  
  reduction<<<1, n>>>(bucket, key);
  hipDeviceSynchronize();
  
  int *tmp;
  hipMallocManaged(&tmp, range*sizeof(int));
  makeoffset<<<1, range>>>(bucket, tmp, range);
  hipDeviceSynchronize();

  sort<<<1, n>>>(bucket, key, range);
  hipDeviceSynchronize();

  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
  hipFree(tmp);
}
